#define N 1000

#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"

void initialize(float A[N][N], float B[N][N]) {
    srand(0);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = rand() / (RAND_MAX * 1.0);
            B[i][j] = rand() / (RAND_MAX * 1.0);
        }
    }
}

void display(float mat[N][N]) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f\t", mat[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}


__global__ void matMul(float *A, float *B, float *C) {
    int block = blockIdx.x;     // Used to index row of matrix
    int thread = threadIdx.x;   // Used to index column of matrix
    
    int dim = blockDim.x;

    float sum = 0;
    for(int i = 0; i < N; i++){
        sum += A[block*dim + i] * B[i*dim + thread];
    }

    C[block*dim + thread] = sum;
}


int main(int argc, char **argv) {
    float A[N][N], B[N][N], C[N][N];

    clock_t start, end;
    double  elapsed;

    float *dev_A, *dev_B, *dev_C;
    int size = N*N*sizeof(float);
    
    hipMalloc((void **)&dev_A, size);
    hipMalloc((void **)&dev_B, size);
    hipMalloc((void **)&dev_C, size);

    initialize(A, B);

    start = clock();

    // Copy inputs from host to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Launch the Matrix multiplication kernel
    matMul<<<N, N>>>(dev_A, dev_B, dev_C);

    // Copy result from device to host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    end = clock();
    elapsed = ((double) (end - start)) / CLOCKS_PER_SEC;

    // display(A);
    // display(B);
    // display(C);

    printf(" \n Time taken is %f \n",elapsed);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}
